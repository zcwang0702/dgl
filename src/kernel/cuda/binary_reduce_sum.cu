/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/binary_reduce_sum.cu
 * \brief CUDA kernels for binary reduce sum
 */
#include <dgl/runtime/device_api.h>

#include "../../runtime/cuda/cuda_common.h"
#include "./binary_reduce_impl.cuh"
#include "./backward_binary_reduce_impl.cuh"
#include "../utils.h"

using minigun::advance::RuntimeConfig;
using Csr = minigun::Csr<int32_t>;

namespace dgl {
namespace kernel {
namespace cuda {
// specialization for cusparse

template <typename DType>
hipsparseStatus_t Xcsrmm2(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const DType* alpha, const hipsparseMatDescr_t descrA,
    const DType* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const DType* B, int ldb, const DType* beta, DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPSPARSE_STATUS_EXECUTION_FAILED;
}

template <>
hipsparseStatus_t Xcsrmm2<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const float* alpha, const hipsparseMatDescr_t descrA,
    const float* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const float* B, int ldb, const float* beta, float* C, int ldc) {
  return hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <>
hipsparseStatus_t Xcsrmm2<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const double* alpha, const hipsparseMatDescr_t descrA,
    const double* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const double* B, int ldb, const double* beta, double* C, int ldc) {
  return hipsparseDcsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <typename DType>
void CusparseCsrmm2(
    const RuntimeConfig& rtcfg,
    const Csr& csr,
    const DType* B_data, DType* C_data,
    int out_size, int x_length) {
  // We use csrmm2 to perform following operation:
  // C = A x B, where A is a sparse matrix in csr format, B is the dense matrix for node
  // feature tensor. However, since cusparse only supports column-major, while our tensor
  // is stored in row-major, the actual computation is:
  // C = trans(A x trans(B)).
  // Currently, we use cublasXgeam to implement transposition and allocate intermediate
  // workspace memory for this.
  // TODO(minjie): The given CSR could potentially represent a bipartite graph (e.g. in the
  //   case of nodeflow). Currently, we don't have bipartite graph support. Here is a small
  //   hack. In the python side, we create a CSR that includes both the source and destination
  //   nodes in the bipartite graph (so it is still square matrix). Here, when multiplying
  //   this sparse matrix, we specify the number of rows (the `m` here) to be equal to the
  //   number of rows of the output tensor (i.e, the `out_size`).
  //   In the future, we should make sure the number of rows of the given csr is equal
  //   to out_size (a.k.a the given csr is a rectangle matrix).
  const int m = out_size;
  const int k = csr.row_offsets.length - 1;
  const int n = x_length;
  const int nnz = csr.column_indices.length;
  const DType alpha = 1.0;
  const DType beta = 0.0;
  // device
  auto device = runtime::DeviceAPI::Get(rtcfg.ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, rtcfg.stream));
  // allocate matrix for temporary transposed output
  DType* trans_out = static_cast<DType*>(device->AllocWorkspace(rtcfg.ctx, m * n * sizeof(DType)));
  // all one data array
  DType* valptr = static_cast<DType*>(device->AllocWorkspace(rtcfg.ctx, nnz * sizeof(DType)));
  utils::Fill<kDLGPU>(rtcfg.ctx, valptr, nnz, static_cast<DType>(1.));
  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  CUSPARSE_CALL(Xcsrmm2<DType>(
      thr_entry->cusparse_handle,
      HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_TRANSPOSE,
      m, n, k, nnz, &alpha,
      descr, valptr, csr.row_offsets.data, csr.column_indices.data,
      B_data, n, &beta, trans_out, m));
  device->FreeWorkspace(rtcfg.ctx, valptr);
  // transpose the output matrix
  if (!thr_entry->cublas_handle) {
    CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
  }
  CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle, rtcfg.stream));
  CUBLAS_CALL(Xgeam<DType>(
      thr_entry->cublas_handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      n, m,
      &alpha, trans_out, m,
      &beta, nullptr, n,
      C_data, n));
  device->FreeWorkspace(rtcfg.ctx, trans_out);
}

// forward

template <typename DType>
void FallbackCallBinaryReduce(
    const RuntimeConfig& rtcfg,
    const ImmutableGraph* graph,
    GData<int32_t, DType>* gdata) {
  constexpr int XPU = kDLGPU;
  typedef int32_t Idx;
  typedef SelectSrc LeftSelector;
  typedef SelectNone RightSelector;
  typedef BinaryUseLhs<DType> BinaryOp;
  typedef ReduceSum<kDLGPU, DType> Reducer;
  typedef cuda::FunctorsTempl<Idx, DType, LeftSelector,
                        RightSelector, BinaryOp, Reducer>
          Functors;
  typedef cuda::BinaryReduce<Idx, DType, Functors> UDF;
  // csr
  auto outcsr = graph->GetOutCSR();
  minigun::Csr<Idx> csr = utils::CreateCsr<Idx>(outcsr->indptr(), outcsr->indices());
  // If the user-given mapping is none and the target is edge data, we need to
  // replace the mapping by the edge ids in the csr graph so that the edge
  // data is correctly read/written.
  if (LeftSelector::target == binary_op::kEdge && gdata->lhs_mapping == nullptr) {
    gdata->lhs_mapping = static_cast<Idx*>(outcsr->edge_ids()->data);
  }
  if (RightSelector::target == binary_op::kEdge && gdata->rhs_mapping == nullptr) {
    gdata->rhs_mapping = static_cast<Idx*>(outcsr->edge_ids()->data);
  }
  if (OutSelector<Reducer>::Type::target == binary_op::kEdge
      && gdata->out_mapping == nullptr) {
    gdata->out_mapping = static_cast<Idx*>(outcsr->edge_ids()->data);
  }
  // TODO(minjie): allocator
  minigun::advance::Advance<XPU, Idx, cuda::AdvanceConfig, GData<Idx, DType>, UDF>(
        rtcfg, csr, gdata, minigun::IntArray1D<Idx>());
}

template <typename DType>
void FallbackCallBackwardBinaryReduce(
    const RuntimeConfig& rtcfg,
    const ImmutableGraph* graph,
    BackwardGData<int32_t, DType>* gdata) {
  constexpr int XPU = kDLGPU;
  constexpr int Mode = binary_op::kGradLhs;
  typedef int32_t Idx;
  typedef SelectSrc LeftSelector;
  typedef SelectNone RightSelector;
  typedef BinaryUseLhs<DType> BinaryOp;
  typedef ReduceSum<kDLGPU, DType> Reducer;
  // For backward computation, we use reverse csr and switch dst and src.
  // This benefits the most common src_op_edge or copy_src case, because the
  // gradients of src are now aggregated into destination buffer to reduce
  // competition of atomic add.
  auto incsr = graph->GetInCSR();
  minigun::Csr<Idx> csr = utils::CreateCsr<Idx>(incsr->indptr(), incsr->indices());
  typedef cuda::BackwardFunctorsTempl<Idx, DType,
          typename SwitchSrcDst<LeftSelector>::Type,
          typename SwitchSrcDst<RightSelector>::Type,
          BinaryOp, Reducer> Functors;
  typedef cuda::BackwardBinaryReduce<Mode, Idx, DType, Functors> UDF;
  // If the user-given mapping is none and the target is edge data, we need to
  // replace the mapping by the edge ids in the csr graph so that the edge
  // data is correctly read/written.
  if (LeftSelector::target == binary_op::kEdge
      && gdata->lhs_mapping == nullptr) {
    gdata->lhs_mapping = static_cast<Idx*>(incsr->edge_ids()->data);
  }
  if (RightSelector::target == binary_op::kEdge
      && gdata->rhs_mapping == nullptr) {
    gdata->rhs_mapping = static_cast<Idx*>(incsr->edge_ids()->data);
  }
  if (OutSelector<Reducer>::Type::target == binary_op::kEdge
      && gdata->out_mapping == nullptr) {
    gdata->out_mapping = static_cast<Idx*>(incsr->edge_ids()->data);
  }
  // TODO(minjie): allocator
  minigun::advance::Advance<XPU, Idx, cuda::AdvanceConfig, BackwardGData<Idx, DType>, UDF>(
        rtcfg, csr, gdata, minigun::IntArray1D<Idx>());
}

}  // namespace cuda

template <>
void CallBinaryReduce<kDLGPU, int32_t, float, SelectSrc, SelectNone,
                      BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(
    const RuntimeConfig& rtcfg,
    const ImmutableGraph* graph,
    GData<int32_t, float>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBinaryReduce<float>(rtcfg, graph, gdata);
  } else {
    // cusparse use rev csr for csrmm
    auto incsr = graph->GetInCSR();
    Csr csr = utils::CreateCsr<int32_t>(incsr->indptr(), incsr->indices());
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->lhs_data, gdata->out_data,
        gdata->out_size, gdata->x_length);
  }
}

template <>
void CallBinaryReduce<kDLGPU, int32_t, double, SelectSrc, SelectNone,
                      BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(
    const RuntimeConfig& rtcfg,
    const ImmutableGraph* graph,
    GData<int32_t, double>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBinaryReduce<double>(rtcfg, graph, gdata);
  } else {
    // cusparse use rev csr for csrmm
    auto incsr = graph->GetInCSR();
    Csr csr = utils::CreateCsr<int32_t>(incsr->indptr(), incsr->indices());
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->lhs_data, gdata->out_data,
        gdata->out_size, gdata->x_length);
  }
}

// backward

template <>
void CallBackwardBinaryReduce<kDLGPU, binary_op::kGradLhs, int32_t, float,
                              SelectSrc, SelectNone,
                              BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(
    const RuntimeConfig& rtcfg,
    const ImmutableGraph* graph,
    BackwardGData<int32_t, float>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBackwardBinaryReduce<float>(rtcfg, graph, gdata);
  } else {
    auto outcsr = graph->GetOutCSR();
    Csr csr = utils::CreateCsr<int32_t>(outcsr->indptr(), outcsr->indices());
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->grad_out_data, gdata->grad_lhs_data,
        gdata->out_size, gdata->x_length);
  }
}

template <>
void CallBackwardBinaryReduce<kDLGPU, binary_op::kGradLhs, int32_t, double,
                              SelectSrc, SelectNone,
                              BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(
    const RuntimeConfig& rtcfg,
    const ImmutableGraph* graph,
    BackwardGData<int32_t, double>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBackwardBinaryReduce<double>(rtcfg, graph, gdata);
  } else {
    auto outcsr = graph->GetOutCSR();
    Csr csr = utils::CreateCsr<int32_t>(outcsr->indptr(), outcsr->indices());
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->grad_out_data, gdata->grad_lhs_data,
        gdata->out_size, gdata->x_length);
  }
}

// generate definitions

#define REDUCER ReduceSum
#define XPU kDLGPU
#define IDX int32_t

EVAL(GEN_DTYPE, GEN_OP_TARGET, GEN_DEFINE);
EVAL(GEN_BACKWARD_MODE, GEN_DTYPE, GEN_OP_TARGET, GEN_BACKWARD_DEFINE);

}  // namespace kernel
}  // namespace dgl
